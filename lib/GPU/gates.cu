#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <chrono>
#include "gates.cuh"

using namespace std;
using namespace redcufhe;

void CtxtCopyH2D(const Ctxt& c, Stream st) {
  hipMemcpyAsync(c.lwe_sample_device_->data(),
                  c.lwe_sample_->data(),
                  c.lwe_sample_->SizeData(),
                  hipMemcpyHostToDevice,
                  st.st());
}

void CtxtCopyD2H(const Ctxt& c, Stream st) {
  hipMemcpyAsync(c.lwe_sample_->data(),
                  c.lwe_sample_device_->data(),
                  c.lwe_sample_->SizeData(),
                  hipMemcpyDeviceToHost,
                  st.st());
}

__device__
void NoiselessTrivial(LWESample* result, Torus mu){
    const int32_t n = result->n();
    for (int32_t i = 0; i < n; ++i) result->a()[i] = 0;
    result->b() = mu;
}

__device__
void Copy_leveled(LWESample** out,
          const LWESample* in) {
  for (int i = 0; i <= in->n(); i ++)
    (*out)->data()[i] = in->data()[i];
}

__device__ inline
uint32_t ModSwitch2048(uint32_t a) {
  return (((uint64_t)a << 32) + (0x1UL << 52)) >> 53;
}

__global__
void NandOp(Torus* out, Torus* in0, Torus* in1, uint32_t n, const Torus fix) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = 0 - in0[i] - in1[i];
  }
  if (i == n) {
    out[i] += fix;
  }
}

__global__
void OrOp(Torus* out, Torus* in0, Torus* in1, uint32_t n, const Torus fix) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = 0 + in0[i] + in1[i];
  }
  if (i == n) {
    out[i] += fix;
  }
}

__global__
void AndOp(Torus* out, Torus* in0, Torus* in1, uint32_t n, const Torus fix) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = 0 + in0[i] + in1[i];
  }
  if (i == n) {
    out[i] += fix;
  }
}

__global__
void NorOp(Torus* out, Torus* in0, Torus* in1, uint32_t n, const Torus fix) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = 0 - in0[i] - in1[i];
  }
  if (i == n) {
    out[i] += fix;
  }
}

__global__
void XorOp(Torus* out, Torus* in0, Torus* in1, uint32_t n, const Torus fix) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = 0 + 2*(in0[i] + in1[i]);
  }
  if (i == n) {
    out[i] += fix;
  }
}

__global__
void XnorOp(Torus* out, Torus* in0, Torus* in1, uint32_t n, const Torus fix) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = 0 - 2*in0[i] - 2*in1[i];
  }
  if (i == n) {
    out[i] += fix;
  }
}

__global__
void NotOp(Torus* out, const Torus* in, uint32_t n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = -in[i];
  }
}

void levelNOT(Ctxt& out,
          const Ctxt& in0,
          Stream st) {
  NotOp<<<1,512,0,st.st()>>>(out.lwe_sample_device_->data(), in0.lwe_sample_device_->data(), out.lwe_sample_device_->n());
}

void redsec_binarize_bootstrap(Ctxt& out, Stream st) {
  static const Torus mu = ModSwitchToTorus(-1,8);
  CtxtCopyH2D(out, st);
  Bootstrap(out.lwe_sample_device_, out.lwe_sample_device_, mu, st.st());
  CtxtCopyD2H(out, st);
  CuCheckError();
}

void redsec_unbinarize_bootstrap(Ctxt& out, Stream st) {
  static const Torus mu = ModSwitchToTorus(1, MSG_SPACE);
  CtxtCopyH2D(out, st);
  Bootstrap(out.lwe_sample_device_, out.lwe_sample_device_, mu, st.st());
  CtxtCopyD2H(out, st);
}

void redsec_unbinarize_bootstrap_inv(Ctxt& out, Stream st) {
  static const Torus mu = ModSwitchToTorus(-1, MSG_SPACE);
  CtxtCopyH2D(out, st);
  Bootstrap(out.lwe_sample_device_, out.lwe_sample_device_, mu, st.st());
  CtxtCopyD2H(out, st);
}

void NoiselessTrivial(Ctxt& result, Torus mu){
    const int32_t n = result.lwe_sample_->n();

    for (int32_t i = 0; i < n; ++i) result.lwe_sample_->a()[i] = 0;
    result.lwe_sample_->b() = mu;
}

void levelCONSTANT(Ctxt& result, int32_t value) {
    static const Torus MU = ModSwitchToTorus(1, 8);
    NoiselessTrivial(result, value ? MU : -MU);
}

__global__
void AddOp(Torus* out, Torus* in0, Torus* in1, uint32_t n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = in0[i] + in1[i];
  }
}

__global__
void SubOp(Torus* out, Torus* in0, Torus* in1, uint32_t n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] = in0[i] - in1[i];
  }
}

void mul_int(redcufhe::Ctxt& prod, const redcufhe::Ctxt& a, uint16_t b) {
  for (int i = 0; i < a.lwe_sample_->n(); i++) {
    prod.lwe_sample_->data()[i] = a.lwe_sample_->data()[i]*b;
  }
  prod.lwe_sample_->b() = a.lwe_sample_->b()*b;
  CuCheckError();
}

void add_int(Ctxt& sum, const Ctxt& a, const Ctxt& b, Stream st) {
    CtxtCopyH2D(a, st);
    CtxtCopyH2D(b, st);

    int numBlocks = (sum.lwe_sample_->n() + 512 - 1)/512;

    AddOp<<<numBlocks,512,0,st.st()>>>(sum.lwe_sample_device_->data(), a.lwe_sample_device_->data(),
        b.lwe_sample_device_->data(), sum.lwe_sample_device_->n());

    CtxtCopyD2H(sum, st);
    CuCheckError();
}

void sub_int(Ctxt& res, const Ctxt& a, const Ctxt& b, Stream st) {
    CtxtCopyH2D(a, st);
    CtxtCopyH2D(b, st);

    int numBlocks = (res.lwe_sample_->n() + 512 - 1)/512;

    SubOp<<<numBlocks,512,0,st.st()>>>(res.lwe_sample_device_->data(), a.lwe_sample_device_->data(),
        b.lwe_sample_device_->data(), res.lwe_sample_device_->n());

    CtxtCopyD2H(res, st);
    CuCheckError();
}

void bootstrapped_full_adder(Ctxt& sum, Ctxt& carry_out, Ctxt& temp_a, Ctxt& temp_b, const Ctxt& a, const Ctxt& b, const Ctxt& carry_in, Stream st) {
    // transfer to GPU
    CtxtCopyH2D(temp_a, st);
    CtxtCopyH2D(temp_b, st);
    CtxtCopyH2D(a, st);
    CtxtCopyH2D(b, st);
    CtxtCopyH2D(carry_in, st);

    // temp[0] = a XOR b
    static const Torus fix = ModSwitchToTorus(1, 4);
    static const Torus mu = ModSwitchToTorus(1, 8);
    XorBootstrap(temp_a.lwe_sample_device_, a.lwe_sample_device_,
        b.lwe_sample_device_, mu, fix, st.st());


    // sum = temp[0] XOR carry_in
    XorBootstrap(sum.lwe_sample_device_, temp_a.lwe_sample_device_,
        carry_in.lwe_sample_device_, mu, fix, st.st());

    // temp[0] = carry_in AND temp[0]
    static const Torus fix_and = ModSwitchToTorus(-1, 8);
    AndBootstrap(temp_a.lwe_sample_device_, carry_in.lwe_sample_device_,
        temp_a.lwe_sample_device_, mu, fix_and, st.st());

    // temp[1] = a AND b
    AndBootstrap(temp_b.lwe_sample_device_, a.lwe_sample_device_,
        b.lwe_sample_device_, mu, fix_and, st.st());

    // carry_out = temp[0] OR temp[1]
    static const Torus fix_or = ModSwitchToTorus(1, 8);
    OrBootstrap(carry_out.lwe_sample_device_, temp_a.lwe_sample_device_,
        temp_b.lwe_sample_device_, mu, fix_or, st.st());

    // transfer to CPU
    CtxtCopyD2H(sum, st);
    CtxtCopyD2H(carry_out, st);
}

void bootsNAND(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 8);
  NandBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
}

void bootsOR(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 8);
  OrBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
}

void bootsAND(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 8);
  AndBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
}

void bootsNOR(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 8);
  NorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
}

void bootsXOR(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 4);
  XorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
}

void bootsXNOR(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 4);
  XnorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
}

void deviceCopy(Ctxt& out,
          const Ctxt& in,
          Stream st) {
  for (int i = 0; i <= in.lwe_sample_device_->n(); i ++)
    out.lwe_sample_device_->data()[i] = in.lwe_sample_device_->data()[i];
}
