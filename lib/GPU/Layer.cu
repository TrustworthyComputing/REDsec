#include<cstdint>
#include<cstdlib>
#include<cstdio>
#include "Layer.cuh"

void print_status(const char* s)
{
#ifdef _PRINT_STATUS_
    printf("%s", s) ;
#endif
}

uint64_t get_size(tRectangle* ws, uint16_t in_dep, uint16_t out_dep)
{
    return (ws->h) * (ws->w) * (in_dep) * (out_dep)  ;
}

void netParamsCpy(tNetParams* dest, tNetParams* src)
{
    memcpy(&(dest->conv), &(src->conv), sizeof(tConvParams)) ;
    memcpy(&(dest->pool), &(src->pool), sizeof(tPoolParams)) ;
    memcpy(&(dest->bnorm), &(src->bnorm), sizeof(tBNormParams)) ;
    dest->e_bias = src->e_bias ;
    dest->version = src->version ;
}

void* arr_calloc(uint32_t len, uint8_t type_size)
{
    return calloc(len, type_size) ;
}

void bit_calloc(tBit** ret, uint32_t len)
{
    *ret = new redcufhe::Ctxt[len];
    return;
}

void bit_calloc_global(tBitPacked** ret, uint32_t len)
{
    omp_set_num_threads(NUM_GPUS);
    *ret = new tBitPacked;
    #pragma omp parallel for shared(ret)
    for (int i = 0; i < NUM_GPUS; i++) {
      hipSetDevice(i);
      (*ret)->enc_segs[i] = new redcufhe::Ctxt[len];
    }
    (*ret)->size = (uint8_t) len;
    return;
}

void mbit_calloc(tMultiBit** ret, uint32_t len, uint8_t bits)
{
    *ret = new tMultiBit[len];
    for (uint32_t i = 0; i < len; i++)
    {
      (*ret)[i].size = bits;
      (*ret)[i].ctxt = new redcufhe::Ctxt[(*ret)[i].size];
    }
    return;
}

void mbit_calloc_global(tMultiBitPacked** ret, uint32_t len, uint8_t bits)
{
    omp_set_num_threads(NUM_GPUS);
    *ret = new tMultiBitPacked;
    (*ret)->size = (uint8_t) len;
    #pragma omp parallel for shared(ret)
    for (int i = 0; i < NUM_GPUS; i++) {
      hipSetDevice(i);
      (*ret)->enc_segs[i] = new tMultiBit[len];
      for (uint32_t j = 0; j < len; j++) {
        (*ret)->enc_segs[i][j].size = bits;
        (*ret)->enc_segs[i][j].ctxt = new redcufhe::Ctxt[bits];
      }
    }
    return;
}

void fixpt_calloc(tFixedPoint** ret, uint32_t len, uint8_t bits)
{
    *ret = new tFixedPoint[len];
    for (uint32_t i = 0; i < len; i++)
    {
      (*ret)[i].size = 1;
      (*ret)[i].ctxt = new redcufhe::Ctxt[1];
    }
    return;
}

void fixpt_calloc_global(tFixedPointPacked** ret, uint32_t len, uint8_t bits)
{
    omp_set_num_threads(NUM_GPUS);
    *ret = new tFixedPointPacked;
    (*ret)->size = (uint8_t) len;
    #pragma omp parallel for shared(ret)
    for (int i = 0; i < NUM_GPUS; i++) {
      hipSetDevice(i);
      (*ret)->enc_segs[i] = new tFixedPoint[len];
      for (uint32_t j = 0; j < len; j++) {
        (*ret)->enc_segs[i][j].size = bits;
        (*ret)->enc_segs[i][j].ctxt = new redcufhe::Ctxt[bits];
      }
    }
    return;
}

void bit_free(uint32_t len, tBit* to_free)
{
    delete [] to_free;
}

void bit_free_global(tBitPacked* to_free) {
  omp_set_num_threads(NUM_GPUS);
  #pragma omp parallel for shared(to_free)
  for (int i = 0; i < NUM_GPUS; i++) {
    hipSetDevice(i);
    delete [] to_free->enc_segs[i];
  }
  delete to_free;
}

void mbit_free(uint32_t len, tMultiBit* to_free)
{
    for (uint32_t i = 0; i < len; i++)
    {
      delete [] to_free[i].ctxt;
    }
    delete to_free;
}

void mbit_free_global(uint32_t len, tMultiBitPacked* to_free) {
  omp_set_num_threads(NUM_GPUS);
  #pragma omp parallel for shared(to_free)
  for (int i = 0; i < NUM_GPUS; i++) {
    hipSetDevice(i);
    for (uint32_t j = 0; j < len; j++) {
      delete [] to_free->enc_segs[i][j].ctxt;
    }
    delete to_free->enc_segs[i];
  }
  delete to_free;
}

void fixpt_free(uint32_t len, tFixedPoint* to_free)
{
    for (uint32_t i = 0; i < len; i++)
    {
      delete [] to_free[i].ctxt;
    }
    delete to_free;
}

void fixpt_free_global(uint32_t len, tMultiBitPacked* to_free) {
  omp_set_num_threads(NUM_GPUS);
  #pragma omp parallel for shared(to_free)
  for (int i = 0; i < NUM_GPUS; i++) {
    hipSetDevice(i);
    for (uint32_t j = 0; j < len; j++) {
      delete [] to_free->enc_segs[i][j].ctxt;
    }
    delete to_free->enc_segs[i];
  }
  delete to_free;
}
