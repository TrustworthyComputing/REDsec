#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdlib>
#include <cassert>
#include <cstring>
#include <string>
#include <chrono>
#include <iostream>
#include "Layer.cuh"
#include <omp.h>
#include <cmath>
#include "BinFunc_gpu.cuh"
#include "BinOps_gpu.cuh"
#include "IntOps_gpu.cuh"

using namespace std;
using namespace std::chrono;
using namespace redcufhe;

BinFunc::Convolution::Convolution(uint32_t out_dep, tConvParams* in_params)
{
    //error checking
    assert(in_params != NULL) ;
    assert(out_dep > 0) ;
    assert((in_params->window.h > 0) && (in_params->window.w > 0));

    OutDepth = out_dep ;
    memcpy(&conv, in_params, sizeof(conv)) ;
    b_prep = false ;
}

tDimensions* BinFunc::Convolution::prep(FILE* fd_filt, tDimensions* ret_dim)
{
    assert(!b_prep) ;
    assert((ret_dim != NULL) && (fd_filt != NULL)) ;
    assert((conv.stride.h != 0) && (conv.stride.w != 0)) ;

    //set parameters depth
    memcpy(&lay_dim, ret_dim, sizeof(lay_dim)) ;

    if(conv.same_pad)
    {   //zero padding -offset is half of window size
        out_hw.h = (lay_dim.hw.h-1)/conv.stride.h + 1 ;
        out_hw.w = (lay_dim.hw.w-1)/conv.stride.w + 1 ;
        if(conv.stride.h == 1){ offset_window.h = (int16_t)((conv.window.h-1)/2) ; }
	    else{ offset_window.h = (out_hw.h*conv.stride.h - lay_dim.hw.h)/2 ; }
        if(conv.stride.w == 1){ offset_window.w = (int16_t)((conv.window.w-1)/2) ; }
	    else{ offset_window.w = (out_hw.w*conv.stride.w - lay_dim.hw.w)/2 ; }
    }
    else //valid padding - no offest, but remove left and right borders
    {
        offset_window.w = 0 ;
        offset_window.h = 0 ;
        out_hw.h = lay_dim.hw.h - 2*((int16_t)((conv.window.h-1)/2)) ;
        out_hw.h = out_hw.h/conv.stride.h ;
        out_hw.w = lay_dim.hw.w - 2*((int16_t)((conv.window.w-1)/2)) ;
        out_hw.w = out_hw.w/conv.stride.w ;
    }

    //calculate number of additions for output bitsize
    lay_dim.up_bound *=
        (lay_dim.filter_bits) *
        (conv.window.w) *
        (conv.window.h) *
	      (lay_dim.in_dep) ;
    for(lay_dim.out_bits = lay_dim.in_bits ;
        (lay_dim.up_bound >> lay_dim.out_bits) > 0 ;
        lay_dim.out_bits++) ;

    //allocate memory
    flen = get_size(&conv.window, lay_dim.in_dep, OutDepth) ;
    p_filters = new uint8_t [flen] ;
    p_tern = new uint8_t [flen] ;

    //get filters
    BinOps::get_ternfilters(fd_filt, p_filters, p_tern, flen, conv.tern_thresh) ;

    //update dimensions
    ret_dim->hw.h = out_hw.h ;
    ret_dim->hw.w = out_hw.w ;
    ret_dim->in_dep = OutDepth ;
    ret_dim->in_bits = lay_dim.out_bits  ;
    ret_dim->up_bound = lay_dim.up_bound  ;
    ret_dim->scale = lay_dim.scale ;
    ret_dim->out_bits = SINGLE_BIT ;

    b_prep = true ;
    uint32_t partsum_ops = (conv.window.h)*(conv.window.w)*(lay_dim.in_dep) ;

    mbit_calloc_global(&p_window, partsum_ops, 1);
    mbit_calloc_global(&p_output, (lay_dim.hw.h)*(lay_dim.hw.w)*OutDepth, 1) ;
    bit_calloc_global(&p_inputs_bar, (lay_dim.hw.h)*(lay_dim.hw.w)*(lay_dim.in_dep));

    return ret_dim ;
}

tMultiBitPacked* BinFunc::Convolution::execute(tBitPacked* p_inputs)
{
    uint64_t input_i, filt_i, output_i ;
    uint8_t oob_i;

    //shorter names for offset, convolution windows
    int16_t ofs_h = offset_window.h ;
    int16_t ofs_w = offset_window.w ;

    //number of items per output
    uint32_t partsum_ops = (conv.window.h)*(conv.window.w)*(lay_dim.in_dep) ;

    //make sure model was prepared
    assert(b_prep) ;

    uint32_t od, ph, pw, fh, fw, di, wi;
    bool oob;
    int idx;
    uint64_t ops;
    int st_ctr = 0;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    #pragma omp parallel for collapse(3) shared(p_output) private(st, st_ctr, idx, flag, input_i, filt_i, output_i, ph, pw, fh, fw, di, oob, ops)
    for(ph = 0 ; ph < lay_dim.hw.h ; ph++)
    {
        for(pw = 0 ; pw < lay_dim.hw.w ; pw++)
        {
            for(di = 0 ; di < lay_dim.in_dep ; di++)
            {
                if (flag != 1) { // should execute once per CPU thread
                    idx = omp_get_thread_num();
                    hipSetDevice(idx);
                    st_ctr = 0;
                    st[idx] = new Stream[sm_num];
                    for (int i = 0; i < sm_num; i++) {
                        st[idx][i].Create();
                    }
                    flag = 1;
                    Synchronize();
                }
                //Loop 1a: multiply
                assert(p_inputs_bar != NULL) ;
                input_i = get_input_i(ph, pw, di) ;
                //XNOR 0 is NOT
                BinOps::multiply(&p_inputs_bar->enc_segs[idx][input_i], &p_inputs->enc_segs[idx][input_i], 0, st[idx][st_ctr % sm_num]) ;
                //XNOR 1 is copy - use original input
                BinOps::unbinarize_int_inv(p_inputs_bar->enc_segs[idx][input_i], st[idx][st_ctr % sm_num]);
                BinOps::unbinarize_int(p_inputs->enc_segs[idx][input_i], st[idx][st_ctr % sm_num]);
                st_ctr++;
            }
        }
    }
    Synchronize();
    flag = 0;
    #pragma omp parallel for collapse(3) private(st_ctr, wi, idx, flag, input_i, oob_i, filt_i, output_i, fh, fw, di, oob, ops)
    for(od = 0 ; od < OutDepth ; od++)
    {   //output picture dimensions
        for(ph = 0 ; ph < out_hw.h ; ph++)
        {
            for(pw = 0 ; pw < out_hw.w ; pw++)
            {
                if (flag != 1) { // should execute once per CPU thread
                  idx = omp_get_thread_num();
                  hipSetDevice(idx);
                  st_ctr = 0;
                  st[idx] = new Stream[sm_num];
                  for (int i = 0; i < sm_num; i++) {
                    st[idx][i].Create();
                  }
                  flag = 1;
                  Synchronize();
                }
		        oob_i = 0;
                for(wi = 0 ; wi < partsum_ops ; wi++)
                {
                    oob = retrieve_dims(wi, ph, pw, &di, &fh, &fw) ;
                    filt_i = get_filter_i(fh, fw, di, od) ;
                    input_i = get_input_i((fh+ph-ofs_h), (fw+pw-ofs_w), di) ;
                    if(!oob && ((p_tern==NULL) || (p_tern[filt_i] == 0)))
                    {
                        if(p_filters[filt_i] == 0)
                        {
                            for (int el = 0; el <= p_window->enc_segs[idx][wi].ctxt[0].lwe_sample_->n(); el ++)
                              p_window->enc_segs[idx][wi].ctxt[0].lwe_sample_->data()[el] = p_inputs_bar->enc_segs[idx][input_i].lwe_sample_->data()[el];
                        }
                        else
                        {
                            for (int el = 0; el <= p_window->enc_segs[idx][wi].ctxt[0].lwe_sample_->n(); el ++)
                              p_window->enc_segs[idx][wi].ctxt[0].lwe_sample_->data()[el] = p_inputs->enc_segs[idx][input_i].lwe_sample_->data()[el];
                        }
                    }
                    //padding, alternate 0s and 1s
                    else if(p_tern != NULL && p_tern[filt_i] == 1)
                    {
                       Torus mu = ModSwitchToTorus(-1, MSG_SPACE);
                       NoiselessTrivial(p_window->enc_segs[idx][wi].ctxt[0], mu);
                    }
                    else {
                        if(((oob_i++) % 2) == 0)
                        {
                            Torus mu = ModSwitchToTorus(-1, MSG_SPACE);
                            NoiselessTrivial(p_window->enc_segs[idx][wi].ctxt[0], mu);
                        }
                        else
                        {
                            Torus mu = ModSwitchToTorus(1, MSG_SPACE);
                            NoiselessTrivial(p_window->enc_segs[idx][wi].ctxt[0], mu);
                        }
                    }
                    //zero weight value, handled in bias
                }
                output_i = get_output_i(ph, pw, od) ;
                for (ops = 0; ops < partsum_ops; ops++) {
                  BinOps::int_add(p_output->enc_segs[idx][output_i].ctxt[0], p_output->enc_segs[idx][output_i].ctxt[0], p_window->enc_segs[idx][ops].ctxt[0], st[idx][st_ctr % sm_num]);
                  st_ctr = (st_ctr + 1) % 40;
                }
            }
        }
    }
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "BinConv: " << time_span.count() << " seconds." << std::endl;

    //free constants input data
    bit_free_global(p_inputs) ;
    bit_free_global(p_inputs_bar);
    mbit_free_global(partsum_ops, p_window);
    return p_output ;
}

bool inline BinFunc::Convolution::retrieve_dims(uint64_t i, uint32_t ph, uint32_t pw,
                                        uint32_t* di, uint32_t* fh, uint32_t* fw)
{
    bool oob = false ;
    uint32_t cw_area = conv.window.h * conv.window.w ;
    *di = i/cw_area ;
    *fh = (i%cw_area)/conv.window.w  ;
    *fw = i%conv.window.w  ;
    //bounds check 1: make sure it is in bounds of num_ops
    if(i >= (lay_dim.in_dep*cw_area)){ oob = true ; }
    //bounds check 2: for same padding, make sure filter is in bounds
    if(conv.same_pad)
    {
        if(((uint32_t)((*fh)+ph*conv.stride.h-offset_window.h) >= lay_dim.hw.h) ||
            ((uint32_t)((*fw)+pw*conv.stride.w-offset_window.w) >= lay_dim.hw.w))
        {  oob = true ; }
    }
    return oob ;
}

uint64_t inline BinFunc::Convolution::get_input_i(uint32_t ph, uint32_t pw, uint32_t di)
{
    return (((ph)*lay_dim.hw.w + pw)*lay_dim.in_dep + di) ;
}
uint64_t inline BinFunc::Convolution::get_filter_i(uint32_t fh, uint32_t fw, uint32_t di, uint32_t od)
{
    return ((((fh)*conv.window.w + fw)*lay_dim.in_dep + di)*OutDepth + od) ;
}
uint64_t inline BinFunc::Convolution::get_output_i(uint32_t ph, uint32_t pw, uint32_t od)
{
    return (((ph)*out_hw.w + pw)*OutDepth + od) ;
}
//used for debugging
void BinFunc::Convolution::get_outhw(tRectangle* ret_hw)
{
    memcpy(ret_hw, &out_hw, sizeof(out_hw));
}
void BinFunc::Convolution::get_outdep(uint32_t* out_dep)
{
    *out_dep = OutDepth ;
}

//these functions should only be used by weight_convert program
void BinFunc::Convolution::extract_bias(FILE* fd_filt, tMultiBitPacked* p_bias, eBiasType e_bias){ printf("Weight convert not defined\r\n") ; }
void BinFunc::Convolution::export_weights(FILE* fd){ printf("Weight convert not defined\r\n") ; }

BinFunc::SumPooling::SumPooling(tPoolParams* in_params)
{
    //error checking
    assert(in_params != NULL) ;
    assert((in_params->window.h > 0) && (in_params->window.w > 0));

    memcpy(&pool, in_params, sizeof(pool)) ;
    if(pool.stride.h == 0){ pool.stride.h = pool.window.h ; }
    if(pool.stride.w == 0){ pool.stride.w = pool.window.w ; }
    b_prep = false ;
}


tDimensions* BinFunc::SumPooling::prep(tDimensions* ret_dim)
{
    //input error checking
    assert(!b_prep) ;
    assert((ret_dim != NULL)) ;
    assert(((pool.window.h) != 0) && ((pool.window.w) != 0)) ;
    assert((pool.stride.h) != 0 && pool.stride.w != 0) ;

    //copy dimension
    memcpy(&lay_dim, ret_dim, sizeof(lay_dim)) ;

    //set output depth
    if(pool.same_pad) //round up - (partial pools used)
    {
        out_hw.h = (lay_dim.hw.h - 1)/(pool.stride.h) + 1 ;
        out_hw.w = (lay_dim.hw.w - 1)/(pool.stride.w) + 1 ;
        if(pool.stride.h == 1){ offset_window.h = (int16_t)((pool.window.h-1)/2) ; }
        else{ offset_window.h = (out_hw.h*pool.stride.h - lay_dim.hw.h)/2 ; }
        if(pool.stride.w == 1){ offset_window.w = (int16_t)((pool.window.w-1)/2) ; }
        else{ offset_window.w = (out_hw.w*pool.stride.w - lay_dim.hw.w)/2 ; }
    }
    else //valid pad - truncate (partial pools ignored)
    {
        offset_window.h = 0 ;
        offset_window.w = 0 ;
        out_hw.h = (lay_dim.hw.h-((uint16_t)(pool.window.h/2))-1)/pool.stride.h + 1 ;
        out_hw.w = (lay_dim.hw.w-((uint16_t)(pool.window.w/2))-1)/pool.stride.w + 1 ;
    }

    //update upper bound and out bits
    lay_dim.up_bound *= (pool.window.w) * (pool.window.h) ;
    for(lay_dim.out_bits = lay_dim.in_bits ;
        (lay_dim.up_bound >> lay_dim.out_bits) > 0 ;
        lay_dim.out_bits++) ;

    //update return dimensions
    ret_dim->hw.h = out_hw.h ;
    ret_dim->hw.w = out_hw.w ;
    ret_dim->in_bits = lay_dim.out_bits ;
    ret_dim->up_bound = lay_dim.up_bound  ;
    ret_dim->scale = lay_dim.scale * (pool.window.w) * (pool.window.h) ;
    ret_dim->out_bits = SINGLE_BIT ;

    b_prep = true ;
    return ret_dim ;
}

tMultiBitPacked* BinFunc::SumPooling::execute(tMultiBitPacked* p_inputs)
{
    //input picture indexes
    tRectangle ip = {0,0} ;
    uint64_t input_i, output_i ;

    //input error checking
    assert(b_prep) ;

    //allocate memory
    uint32_t len = get_size(&out_hw, SIZE_EMPTY, lay_dim.in_dep) ;
    tMultiBitPacked* p_output;
    mbit_calloc_global(&p_output, len, MULTIBIT_BITS) ;
    uint32_t opw, oph, di;
    uint32_t fh, fw;
    int idx;
    int st_ctr = 0;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for collapse(3) shared(st) private(flag, idx, st_ctr, output_i, opw, oph, di, fh, fw)
    for(di = 0 ; di < lay_dim.in_dep ; di++)
    {
        for(oph = 0 ; oph < out_hw.h ; oph++)
        {
            for(opw = 0 ; opw < out_hw.w; opw++)
            {
                if (flag != 1) { // should execute once per CPU thread
                  idx = omp_get_thread_num();
                  hipSetDevice(idx);
                  st_ctr = -1;
                  st[idx] = new Stream[sm_num];
                  for (int i = 0; i < sm_num; i++) {
                    st[idx][i].Create();
                  }
                  flag = 1;
                }
                //get input picture indexes
                ip.h = oph * (pool.stride.h) - offset_window.h ;
                ip.w = opw * (pool.stride.w) - offset_window.w ;
                output_i = get_output_i(oph, opw, di) ;
                st_ctr++;
                for(fh = 0 ; (fh < (pool.window.h))
                    && ((ip.h+fh) < (lay_dim.hw.h)) ; fh++)
                {
                    if((ip.h + fh) < 0){ continue ; }
                    for(fw = 0 ; (fw < (pool.window.w))
                        && ((ip.w+fw) < lay_dim.hw.w) ; fw++)
                    {
                     	if((ip.w + fw) < 0){ continue ; }
                        input_i = get_input_i(ip.h+fh, ip.w+fw, di) ;
                        BinOps::int_add(p_output->enc_segs[idx][output_i].ctxt[0], &p_output->enc_segs[idx][output_i].ctxt[0], &p_inputs->enc_segs[idx][input_i].ctxt[0], st[idx][st_ctr % sm_num]) ;
                    }
                }
            }
        }
    }

    mbit_free_global((lay_dim.hw.h * lay_dim.hw.w * lay_dim.in_dep), p_inputs) ;
    return p_output ;
}

uint64_t inline BinFunc::SumPooling::get_input_i(uint32_t ph, uint32_t pw, uint32_t di)
{
    return (((ph)*lay_dim.hw.w + pw)*lay_dim.in_dep + di) ;
}
uint64_t inline BinFunc::SumPooling::get_output_i(uint32_t ph, uint32_t pw, uint32_t od)
{
    //indepth == outdepth
    return (((ph)*out_hw.w + pw)*lay_dim.in_dep + od) ;
}

void BinFunc::SumPooling::get_outhw(tRectangle* ret_hw)
{
    memcpy(ret_hw, &out_hw, sizeof(out_hw));
}
void BinFunc::SumPooling::get_outdep(uint32_t* out_dep)
{
    *out_dep = lay_dim.in_dep ;
}

//these functions should only be used by weight_convert program
void BinFunc::SumPooling::extract_bias(tMultiBitPacked* p_bias){ printf("Weight convert not defined\r\n") ; }

BinFunc::MaxPooling::MaxPooling(tPoolParams* in_params)
{
    //error checking
    assert(in_params != NULL) ;
    assert((in_params->window.h > 0) && (in_params->window.w > 0));

    memcpy(&pool, in_params, sizeof(pool)) ;
    if(pool.stride.h == 0){ pool.stride.h = pool.window.h ; }
    if(pool.stride.w == 0){ pool.stride.w = pool.window.w ; }
    b_prep = false ;
}

tDimensions* BinFunc::MaxPooling::prep(tDimensions* ret_dim)
{
    //input error checking
    assert(!b_prep) ;
    assert((ret_dim != NULL)) ;
    assert(((pool.window.h) != 0) && ((pool.window.w) != 0)) ;
    assert((pool.stride.h) != 0 && pool.stride.w != 0) ;

    //copy dimension
    memcpy(&lay_dim, ret_dim, sizeof(lay_dim)) ;

    //set output dimensions
    if(pool.same_pad) //round up - (partial pools used)
    {
        out_hw.h = (lay_dim.hw.h - 1)/(pool.stride.h) + 1 ;
        out_hw.w = (lay_dim.hw.w - 1)/(pool.stride.w) + 1 ;
    }
    else //valid pad - truncate (partial pools ignored)
    {
        out_hw.h = (lay_dim.hw.h)/(pool.window.h) ;
        out_hw.w = (lay_dim.hw.w)/(pool.window.w) ;
    }
    //max value stays the same
    lay_dim.out_bits = lay_dim.in_bits ;

    //update dimensions
    ret_dim->hw.h = out_hw.h ;
    ret_dim->hw.w = out_hw.w ;
    ret_dim->in_bits = lay_dim.out_bits ;
    ret_dim->out_bits = SINGLE_BIT ;
    ret_dim->scale = lay_dim.scale ;

    b_prep = true ;

    return ret_dim ;
}

tBitPacked* BinFunc::MaxPooling::execute(tBitPacked* p_inputs)
{
    //input picture indexes
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    tRectangle ip = {0,0} ;
    uint64_t input_i, output_i ;

    //input error checking
    assert(b_prep) ;

    //allocate memory
    uint32_t len = get_size(&out_hw, SIZE_EMPTY, lay_dim.in_dep) ;
    tBitPacked* p_output;
    bit_calloc_global(&p_output, len) ;

    uint32_t opw, oph, di;
    uint32_t fh, fw;
    int idx;
    int st_ctr = 0;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for collapse(3) shared(st, p_inputs, p_output) private(flag, st_ctr, idx, output_i, opw, oph, di, fh, fw)
    for(di = 0 ; di < lay_dim.in_dep ; di++)
    {
        for(oph = 0 ; oph < out_hw.h ; oph++)
        {
            for(opw = 0 ; opw < out_hw.w; opw++)
            {
                if (flag != 1) { // should execute once per CPU thread
                  idx = omp_get_thread_num();
                  hipSetDevice(idx);
                  st_ctr = -1;
                  st[idx] = new Stream[sm_num];
                  for (int i = 0; i < sm_num; i++) {
                    st[idx][i].Create();
                  }
                  flag = 1;
                  Synchronize();
                }
                //get input picture indexes
                ip.h = oph * (pool.stride.h) - offset_window.h ;
                ip.w = opw * (pool.stride.w) - offset_window.w ;
		        output_i = get_output_i(oph, opw, di) ;
                st_ctr++;
                for(fh = 0 ; (fh < (pool.window.h))
                    && ((ip.h+fh) < (lay_dim.hw.h)) ; fh++)
                {
 		            if((ip.h + fh) < 0){ continue ; }
                    for(fw = 0 ; (fw < (pool.window.w))
                        && ((ip.w+fw) < lay_dim.hw.w) ; fw++)
                    {
 		                if((ip.w + fw) < 0){ continue ; }
                        input_i = get_input_i(ip.h+fh, ip.w+fw, di) ;
                        BinOps::max(&p_output->enc_segs[idx][output_i], &p_output->enc_segs[idx][output_i], &p_inputs->enc_segs[idx][input_i], st[idx][st_ctr % sm_num]) ;
                    }
                }
            }
        }
    }
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "BinMaxPool: " << time_span.count() << " seconds.\n";
    bit_free_global(p_inputs) ;
    return p_output ;
}

uint64_t inline BinFunc::MaxPooling::get_input_i(uint32_t ph, uint32_t pw, uint32_t di)
{
    return (((ph)*lay_dim.hw.w + pw)*lay_dim.in_dep + di) ;
}
uint64_t inline BinFunc::MaxPooling::get_output_i(uint32_t ph, uint32_t pw, uint32_t od)
{
    //indepth == outdepth
    return (((ph)*out_hw.w + pw)*lay_dim.in_dep + od) ;
}

 BinFunc::Quantize::Quantize(tQParams* qparam)
{
    assert(qparam!=NULL && qparam->shift_bits > 0);
    dim_len = SIZE_EMPTY ;
    shift_bits = qparam->shift_bits ;
    b_prep = false ;
}
#define SLOPE_BITS 16
tDimensions* BinFunc::Quantize::prep(FILE* fd_bias, tDimensions* ret_dim, tMultiBitPacked* p_bias, uint16_t* p_slope)
{
    //input error checking
    assert(!b_prep) ;
    assert((ret_dim != NULL)) ;

    //get bias offset
    uint32_t bias_len = ret_dim->in_dep ;
    assert((fd_bias != NULL) && (p_bias != NULL)) ;
    BinOps::get_intfilters(fd_bias, &p_bias, bias_len) ;
    if(p_slope != NULL)
    {
	    BinOps::get_intfilters_ptxt(fd_bias, p_slope, bias_len) ;
    }

    //copy dimension
    memcpy(&lay_dim, ret_dim, sizeof(lay_dim)) ;

    uint8_t sb = 0 ;
    for(sb= 0 ; (1<<sb) < sqrt(lay_dim.up_bound)/2 ; sb++);
    slope_bits = SLOPE_BITS + sb ;

    //set output dimensions, other parameters
    lay_dim.out_bits = (shift_bits>1)?(shift_bits+1):1 ;
    dim_len = get_size(&lay_dim.hw, SIZE_EMPTY, lay_dim.in_dep) ;

    //update dimensions
    ret_dim->in_bits = lay_dim.out_bits ;
    ret_dim->out_bits = SINGLE_BIT ;
    ret_dim->up_bound = (1<<(lay_dim.out_bits-1)) ;
    ret_dim->scale = (shift_bits>1) ? (ret_dim->up_bound) : 0.5 ;

    b_prep = true ;

    bit_calloc_global(&p_output, dim_len) ;
    mbit_calloc_global(&x_add, 1, 1) ;

    return ret_dim ;
}

tBitPacked* BinFunc::Quantize::execute(tMultiBitPacked* p_inputs, tMultiBitPacked* p_bias)
{
    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    //input error checking
    assert(b_prep) ;
    uint32_t i;
    int idx;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    int st_ctr = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for firstprivate(st_ctr) private(idx, flag) shared(st, x_add, p_inputs, p_bias, p_output)
    for(i = 0 ; i < dim_len ; i++)
    {
        if (flag != 1) { // should execute once per CPU thread
          idx = omp_get_thread_num();
          hipSetDevice(idx);
          st[idx] = new Stream[sm_num];
          for (int i = 0; i < sm_num; i++) {
            st[idx][i].Create();
          }
          flag = 1;
          Synchronize();
        }
        uint32_t di = i % lay_dim.in_dep ;
        BinOps::int_add(x_add->enc_segs[idx][0].ctxt[0], p_inputs->enc_segs[idx][i].ctxt[0], p_bias->enc_segs[idx][di].ctxt[0], st[idx][st_ctr % sm_num]) ;
        BinOps::binarize_int(x_add->enc_segs[idx][0].ctxt[0], st[idx][st_ctr % sm_num]) ;
        Copy(p_output->enc_segs[idx][i], x_add->enc_segs[idx][0].ctxt[0], st[idx][st_ctr % sm_num]);
        st_ctr++;
    }
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "BinQuant: " << time_span.count() << " seconds." << endl;
    mbit_free_global(dim_len, p_inputs) ;
    mbit_free_global(1, x_add) ;
    return p_output ;
}

//these functions should only be used by weight_convert program
void BinFunc::Quantize::extract_bias(tMultiBitPacked* p_bias, uint16_t* p_slope){ printf("Weight convert not defined\r\n") ; }
void BinFunc::Quantize::export_weights(FILE* fd, tMultiBitPacked* p_bias, uint16_t* p_slope){ printf("Weight convert not defined\r\n") ; }

tMultiBitPacked* BinFunc::Quantize::add_bias(tMultiBitPacked* p_inputs, tMultiBitPacked* p_bias)
{
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    tMultiBitPacked* p_output ;

    //input error checking
    assert(b_prep) ;

    //allocate memory
    mbit_calloc_global(&p_output, dim_len,1) ;
    uint32_t i;
    int idx;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    int st_ctr = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for firstprivate(st_ctr) private(idx, flag) shared(st, p_inputs, p_bias, p_output)
    for(i = 0 ; i < dim_len ; i++)
    {
        if (flag != 1) { // should execute once per CPU thread
          idx = omp_get_thread_num();
          hipSetDevice(idx);
          st[idx] = new Stream[sm_num];
          for (int i = 0; i < sm_num; i++) {
            st[idx][i].Create();
          }
          flag = 1;
          Synchronize();
        }
        uint32_t di = i % lay_dim.in_dep ;
        BinOps::int_add(p_output->enc_segs[idx][i].ctxt[0], p_inputs->enc_segs[idx][i].ctxt[0], p_bias->enc_segs[idx][di].ctxt[0], st[idx][st_ctr % sm_num]) ;
        st_ctr++;
    }
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "BinAddBias: " << time_span.count() << " seconds." << std::endl;
    mbit_free_global(dim_len, p_inputs) ;
    mbit_free_global(1, x_add) ;
    return p_output ;
}


tFixedPointPacked* BinFunc::Quantize::relu_shift(tMultiBitPacked* p_inputs, tMultiBitPacked* p_bias, uint16_t* p_slope)
{
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    assert(b_prep) ;
    int idx;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    int st_ctr = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for firstprivate(st_ctr) private(idx, flag) shared(st, p_inputs, p_bias, p_slope, p_output)
    for(uint32_t i = 0 ; i < dim_len ; i++)
    {
        if (flag != 1) { // should execute once per CPU thread
          idx = omp_get_thread_num();
          hipSetDevice(idx);
          st[idx] = new Stream[sm_num];
          for (int i = 0; i < sm_num; i++) {
            st[idx][i].Create();
          }
          flag = 1;
          Synchronize();
        }
        uint32_t di = i % lay_dim.in_dep ;
	    BinOps::multiply_pc_ints(x_bn->enc_segs[idx][0].ctxt[0], p_inputs->enc_segs[idx][i].ctxt[0], &(p_slope[di]), (lay_dim.in_bits), SLOPE_BITS, st[idx][st_ctr % sm_num]) ;
	    BinOps::int_add(x_fp->enc_segs[idx][0].ctxt[0], x_bn->enc_segs[idx][0].ctxt[0], p_bias->enc_segs[idx][di].ctxt[0], st[idx][st_ctr % sm_num]) ;
        BinOps::binarize_int(x_fp->enc_segs[idx][0].ctxt[0], st[idx][st_ctr % sm_num]) ;
        BinOps::shift(&(x_fp->enc_segs[idx][0]), &(x_fp->enc_segs[idx][0]), slope_bits, shift_bits+1, st[idx][st_ctr % sm_num]) ;
        BinOps::relu(&(p_output_relu->enc_segs[idx][i]), &(x_fp->enc_segs[idx][0]), shift_bits, st[idx][st_ctr % sm_num]) ;
    }

    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "BinReLU: " << time_span.count() << " seconds." << std::endl;
    mbit_free_global(dim_len, p_inputs) ;
    mbit_free_global(1, x_bn) ;

    return p_output_relu ;
}
