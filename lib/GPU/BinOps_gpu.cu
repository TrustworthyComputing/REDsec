#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <cstring>
#include <omp.h>
#include "BinOps_gpu.cuh"

#define ASSERT_BIT
typedef uint8_t bitpack_t ;
typedef enum _DATA_FORMAT
{
    NULL_FMT,
    BIN_FMT,
    TERN_FMT,
    UINT32_FMT,
    INT32_FMT,
    NUM_FMT
} eFmt ;

using namespace std;
using namespace redcufhe;

void BinOps::multiply(tBit* result, const tBit* a, const uint8_t b, Stream curr_sm) {
    if (b == 0) {
      Not(*result, *a, curr_sm);
    }
    else {
      Copy(*result, *a, curr_sm);
    }
}

void BinOps::add_bit(tMultiBit* result, const tBit* a, const tBit* b, Stream curr_sm)
{
    result->ctxt = new Ctxt[2];
    result->size = 2;

    // compute sum
    bootsXOR(result->ctxt[0], *a, *b, curr_sm);

    // compute carry
    bootsAND(result->ctxt[1], *a, *b, curr_sm);
}

void BinOps::add(tMultiBit* result, const tMultiBit* a, const tMultiBit* b, uint8_t bits, Stream curr_sm) {

    uint8_t aligned_size = bits;
    result->size = aligned_size;
    result->ctxt = new Ctxt[result->size];

    tMultiBit carry;
    carry.size = aligned_size;
    carry.ctxt = new Ctxt[carry.size];

    tBit* temp = new Ctxt[3];

    tMultiBit a_aligned;
    a_aligned.size = aligned_size;
    a_aligned.ctxt = new Ctxt[a_aligned.size];

    tMultiBit b_aligned;
    b_aligned.size = aligned_size;
    b_aligned.ctxt = new Ctxt[b_aligned.size];

    for (int i = 0; i < aligned_size; i++) {
      if (i >= a->size)
          levelCONSTANT(a_aligned.ctxt[i], 0);
      else
          Copy(a_aligned.ctxt[i], a->ctxt[i], curr_sm);

      if (i >= b->size)
          levelCONSTANT(b_aligned.ctxt[i], 0);
      else
          Copy(b_aligned.ctxt[i], b->ctxt[i], curr_sm);
    }

    //initialize first carry to 0
    levelCONSTANT(carry.ctxt[0], 0);

    //run full adders
    for (int i = 0; i < (aligned_size - 1); i++) {
      bootstrapped_full_adder(result->ctxt[i], carry.ctxt[i+1], temp[0],
        temp[1], a_aligned.ctxt[i], b_aligned.ctxt[i],
                           carry.ctxt[i], curr_sm);
    }

    bootsXOR(temp[0], a_aligned.ctxt[aligned_size-1], b_aligned.ctxt[aligned_size-1], curr_sm);
    bootsXOR(result->ctxt[aligned_size-1], carry.ctxt[aligned_size-1], temp[0], curr_sm);

    delete [] carry.ctxt;
    delete [] a_aligned.ctxt;
    delete [] b_aligned.ctxt;
    delete [] temp;
}

void BinOps::int_add(Ctxt& result, const Ctxt& a, const Ctxt& b, redcufhe::Stream curr_sm) {
    add_int(result, a, b, curr_sm);
}

void BinOps::multiply_pc_ints(Ctxt& result, Ctxt& in1, const uint16_t* multicand, uint8_t in1_bits, uint8_t in2_bits, redcufhe::Stream curr_sm)
{
    mul_int(result, in1, *multicand);
}

void BinOps::add_pc_ints(Ctxt& result, Ctxt& in1, const uint16_t* addend, uint8_t in_bits, redcufhe::Stream curr_sm)
{
    // Encode plaintext context
    int32_t conv_int = 0;
    conv_int = conv_int + (int32_t)((*addend)&0xFFFF);
    Ctxt* enc_int = new Ctxt[1];
    levelCONSTANT(enc_int[0], conv_int);

    add_int(result, in1, enc_int[0], curr_sm);
    delete [] enc_int;
}

void BinOps::inc(tMultiBit* result, const tMultiBit* a, const tBit* b, Stream curr_sm) {
    tMultiBit carry;
    carry.size = a->size;
    carry.ctxt = new Ctxt[carry.size];
    result->size = a->size;
    result->ctxt = new Ctxt[result->size];

    Copy(carry.ctxt[0], b);

    for (int i = 0; i < (a->size - 1); i++) {
        bootsXOR(result->ctxt[i], carry.ctxt[i], a->ctxt[i], curr_sm);
        bootsAND(carry.ctxt[i+1], carry.ctxt[i], a->ctxt[i], curr_sm);
    }
    bootsXOR(result->ctxt[result->size-1], carry.ctxt[a->size-1], a->ctxt[a->size-1], curr_sm);
    delete [] carry.ctxt;
}

void BinOps::max(tBit* result, const tBit* a, const tBit* b, Stream curr_sm) {
    bootsOR(*result, *a, *b, curr_sm);
}

void BinOps::binarize(tBit* result, const tMultiBit* a) {
    Copy(*result, a->ctxt[a->size - 1]);
}

void BinOps::binarize_int(Ctxt& result, redcufhe::Stream curr_sm) {
    redsec_binarize_bootstrap(result, curr_sm);
}

void BinOps::unbinarize_int_inv(Ctxt& result, redcufhe::Stream curr_sm) {
    redsec_unbinarize_bootstrap_inv(result, curr_sm);
}

void BinOps::unbinarize_int(Ctxt& result, redcufhe::Stream curr_sm) {
    redsec_unbinarize_bootstrap(result, curr_sm);
}

void BinOps::relu(tFixedPoint* result, tMultiBit* in1, uint8_t in_bits, redcufhe::Stream curr_sm)
{
    // check if result is already instantiated
    if (result->size != in_bits) {
      result->size = in_bits;
      result->ctxt = new Ctxt[result->size];
    }

  	//AND each bit with MSB (val.ctxt[in_bits-1])- only keep values greater than threshold
    //do not set top bit, recenter around 0
    for(uint8_t i = 0 ; i < in_bits-1; i++)
    {
      bootsAND(result->ctxt[i], in1->ctxt[i], in1->ctxt[in_bits-1], curr_sm);
    }
}

void BinOps::shift(tMultiBit* result, tMultiBit* in1, uint8_t in_bits, uint8_t shift_bits, redcufhe::Stream curr_sm)
{
    if (result->size != in_bits) {
      result->size = in_bits;
      result->ctxt = new Ctxt[result->size];
    }

    assert((in_bits>0) && (shift_bits <= in_bits));
    	//shift over result to keep values small
    for (int i = 0; i < in_bits; i++) {
      if ((i+shift_bits) > (in_bits-1)) { // sign extend
        Copy(result->ctxt[i], in1->ctxt[in_bits-1]);
      }
      else {
        Copy(result->ctxt[i], in1->ctxt[i+shift_bits]);
      }
    }
}

void BinOps::get_filters(FILE* fd_in, tBitPacked** p_filt_b, uint32_t len)
{
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for shared(p_filt_b)
    for (int i = 0; i < NUM_GPUS; i++) {
      hipSetDevice(i);
      (*p_filt_b)->enc_segs[i] = new redcufhe::Ctxt[len];
    }
    (*p_filt_b)->size = (uint8_t) len;
    float* p_filt_f = (float*) arr_calloc(len, sizeof(float)) ;
    uint8_t* p_filt_u = (uint8_t*) arr_calloc(len, sizeof(uint8_t)) ;

    fread((void*)(p_filt_f), sizeof(float), len, fd_in) ;
    for(uint32_t i = 0 ; i<len ; i++)
    {
        p_filt_u[i] = ((((float*)(p_filt_f))[i]) < 0)?0:1 ;
    }
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for shared(p_filt_b)
    for (int i = 0; i < NUM_GPUS; i++) {
      for (uint32_t j = 0; j < len; j++) {
        hipSetDevice(i);
        levelCONSTANT((*p_filt_b)->enc_segs[i][j], p_filt_u[j]);
      }
    }
    free(p_filt_f) ;
    free(p_filt_u) ;
}

void BinOps::get_ternfilters(FILE* fd_in, uint8_t* p_filt_b, uint8_t* p_filt_tern, uint32_t len, float thresh)
{
    //check data format
    uint8_t version = NULL_FMT ;
    size_t sread = fread(&version, sizeof(uint8_t), 1, fd_in) ;
    assert(version == BIN_FMT || (version == TERN_FMT)) ;
    int NBITS = (version==BIN_FMT)?1:2 ;
    bool b_tern = (version == TERN_FMT) && (p_filt_tern != NULL) ;

    static const uint16_t bits = sizeof(bitpack_t)*8 ;
    uint32_t adj_len = (len*NBITS+bits-1)/bits ;
    bitpack_t* p_filt_pack = new bitpack_t [adj_len] ;

    sread = fread((void*)(p_filt_pack), sizeof(bitpack_t), adj_len, fd_in) ;

    for(uint32_t i = 0 ; i<adj_len ; i++)
    {  //unpack bits
    	for(int32_t j = 0 ;  j <= bits-1 ; j+=NBITS)
    	{
    		if((i*bits+j)/NBITS >= len) { continue ; }
        p_filt_b[(i*bits+j)/NBITS] = ((p_filt_pack[i]>>(bits-j-1)) & 0x1) ;
        if(b_tern){ p_filt_tern[(i*bits+j)/NBITS] = ((p_filt_pack[i]>>(bits-j-2)) & 0x1) ; }
    	}
    }
    if((version != TERN_FMT) && (p_filt_tern != NULL)){ memset(p_filt_tern, 0, len) ; }
    delete p_filt_pack ;
}

void BinOps::get_bitfilters(FILE* fd_in, tBitPacked** p_filt_b, uint32_t len)
{
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for shared(p_filt_b)
    for (int i = 0; i < NUM_GPUS; i++) {
      hipSetDevice(i);
      (*p_filt_b)->enc_segs[i] = new redcufhe::Ctxt[len];
    }
    (*p_filt_b)->size = (uint8_t) len;

    uint16_t bits = sizeof(bitpack_t)*8 ;
    uint32_t adj_len = (len+bits-1)/bits ;
    bitpack_t* p_filt_pack = (bitpack_t*) arr_calloc(adj_len, sizeof(bitpack_t)) ;

    size_t sread = fread((void*)(p_filt_pack), sizeof(bitpack_t), adj_len, fd_in) ;

    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for shared(p_filt_b)
    for (int k = 0; k < NUM_GPUS; k++) {
      for(uint32_t i = 0 ; i<adj_len ; i++)
      {  //unpack bits
      	for(int32_t j = bits-1 ;  j >= 0 ; j--)
      	{
  		if(i*bits+j >= len) { continue ; }
      		levelCONSTANT((*p_filt_b)->enc_segs[k][i*bits+j],(p_filt_pack[i]>>(bits-j-1)) & 0x1) ;
      	}
      }
    }
    free(p_filt_pack) ;
}

void BinOps::get_intfilters(FILE* fd_in, tMultiBitPacked** p_filt_mb, uint32_t len)
{
    uint8_t version = NULL_FMT;
    size_t sread = fread(&version, sizeof(uint8_t), 1, fd_in);
    assert((version == UINT32_FMT) || (version == INT32_FMT));

    int32_t* int_filt = (int32_t*) calloc(len, sizeof(int32_t));

    size_t size = fread((int32_t*)(int_filt), sizeof(int32_t), len, fd_in) ;

    mbit_calloc_global(p_filt_mb, len, 1);
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for shared(p_filt_mb)
    for (int k = 0; k < NUM_GPUS; k++) {
      for (uint32_t i = 0; i < len; i++) {
        Torus mu = ModSwitchToTorus(int_filt[i], MSG_SPACE);
        NoiselessTrivial((*p_filt_mb)->enc_segs[k][i].ctxt[0], mu);
      }
    }
    free(int_filt);
}

void BinOps::get_intfilters_ptxt(FILE* fd_in, uint16_t* p_filt_mb, uint32_t len)
{
    uint8_t version = NULL_FMT;
    size_t sread = fread(&version, sizeof(uint8_t), 1, fd_in);
    assert((version == UINT32_FMT) || (version == INT32_FMT));

    int32_t* int_filt = (int32_t*) calloc(len, sizeof(int32_t));

    size_t size = fread((int32_t*)(int_filt), sizeof(int32_t), len, fd_in) ;

    p_filt_mb = (uint16_t*) calloc(len, sizeof(uint16_t));

    for (uint32_t i = 0; i < len; i++) {
      p_filt_mb[i] = int_filt[i] & 0xFFFF;
    }
    free(int_filt);
}
