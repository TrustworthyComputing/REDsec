#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <assert.h>
#include <cstring>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <ratio>
#include <chrono>
#include <omp.h>
#include <iostream>
#include "IntFunc_gpu.cuh"
#include "Layer.cuh"

#include "IntOps_gpu.cuh"
#include "BinOps_gpu.cuh"

using namespace std;
using namespace std::chrono;
using namespace redcufhe;

IntFunc::Convolution::Convolution(uint16_t out_depth, tConvParams* in_params)
{
	//error checking
    assert(in_params != NULL) ;
    assert(out_depth > 0) ;
    assert((in_params->window.h > 0) && (in_params->window.w > 0));

    OutDepth = out_depth ;
    memcpy(&conv, in_params, sizeof(conv)) ;
    b_prep = false ;
}

tDimensions* IntFunc::Convolution::prep(FILE* fd_filt, tDimensions* ret_dim)
{
    assert(!b_prep) ;
    assert((ret_dim != NULL) && (fd_filt != NULL)) ;
    assert((ret_dim->hw.h >= conv.window.h ) && (ret_dim->hw.w >= conv.window.w)) ;
    assert((conv.stride.h != 0) && (conv.stride.w != 0)) ;
    //set parameters depth
    memcpy(&lay_dim, ret_dim, sizeof(lay_dim)) ;
    if(conv.same_pad)
    {   //zero padding -> offset is half of window size
        out_hw.h = (lay_dim.hw.h - 1)/conv.stride.h + 1 ;
        out_hw.w = (lay_dim.hw.w - 1)/conv.stride.w + 1;
        if(conv.stride.h == 1){ offset_window.h = (int16_t)((conv.window.h-1)/2) ; }
	    else{ offset_window.h = (out_hw.h*conv.stride.h - lay_dim.hw.h)/2 ; }
        if(conv.stride.w == 1){ offset_window.w = (int16_t)((conv.window.w-1)/2) ; }
	    else{ offset_window.w = (out_hw.w*conv.stride.w - lay_dim.hw.w)/2 ; }
    }
    else //valid padding - no offest, but remove left and right borders
    {
        offset_window.h = 0 ;
        offset_window.w = 0 ;
        out_hw.h = lay_dim.hw.h - 2*((int16_t)((conv.window.h-1)/2)) ;
        out_hw.h = (out_hw.h)/conv.stride.h  ;
        out_hw.w = lay_dim.hw.w - 2*((int16_t)((conv.window.w-1)/2)) ;
        out_hw.w = (out_hw.w)/conv.stride.w ;
    }
    //calculate number of additions for output bitsize
    in_up_bound = lay_dim.up_bound;
    lay_dim.up_bound *=
        (lay_dim.filter_bits) *
        (conv.window.w) *
        (conv.window.h) *
        (lay_dim.in_dep) ;

    for(lay_dim.out_bits = lay_dim.in_bits ;
        (lay_dim.up_bound >> lay_dim.out_bits) > 0 ;
        lay_dim.out_bits++) ;

    //allocate memory
    flen = get_size(&conv.window, lay_dim.in_dep, OutDepth) ;
    p_filters = (uint8_t*) calloc(flen, sizeof(uint8_t)) ;
    p_tern = (uint8_t*) calloc(flen, sizeof(uint8_t)) ;

    //get filters
    BinOps::get_ternfilters(fd_filt, p_filters, p_tern, flen, conv.tern_thresh) ;

    //update dimensions
    ret_dim->hw.h = out_hw.h ;
    ret_dim->hw.w = out_hw.w ;
    ret_dim->in_dep = OutDepth ;
    ret_dim->in_bits = lay_dim.out_bits  ;
    ret_dim->out_bits = SINGLE_BIT ; //clear last dimension
    ret_dim->up_bound = lay_dim.up_bound  ;
    ret_dim->scale = lay_dim.scale ;

    uint32_t len = get_size(&out_hw, SIZE_EMPTY, OutDepth) ;
    fixpt_calloc_global(&p_output, len, 1) ;
    b_prep = true ;

    return ret_dim ;
}

tFixedPointPacked* IntFunc::Convolution::execute(tFixedPointPacked* p_inputs)
{
    //shorter names for offset, convolution windows
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    uint8_t ofs_h = offset_window.h ;
    uint8_t ofs_w = offset_window.w ;
    uint8_t cw_h = conv.window.h ;
    uint8_t cw_w = conv.window.w ;
    uint64_t input_i, filt_i, output_i ;

    //make sure model was prepared
    assert(b_prep) ;

    //allocate output memory
    uint32_t len = get_size(&out_hw, SIZE_EMPTY, OutDepth) ;

    uint32_t od;
    uint32_t ph;
    uint32_t pw;
    int idx;
    int sm_ctr;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for collapse(3) shared(st, p_output, p_inputs) private(sm_ctr, idx, flag, input_i, filt_i, output_i, ph, pw)
    for(od = 0 ; od < OutDepth ; od++)
    {    //output picture dimensions
        for(ph = 0 ; ph < out_hw.h ; ph++)
        {
            for(pw = 0 ; pw < out_hw.w ; pw++)
            {
                if (flag != 1) { // should execute once per CPU thread
                  idx = omp_get_thread_num();
                  hipSetDevice(idx);
                  st[idx] = new Stream[sm_num];
                  for (int i = 0; i < sm_num; i++) {
                    st[idx][i].Create();
                  }
                  flag = 1;
                  sm_ctr = -1;
                }
                sm_ctr++;
                output_i = get_output_i(ph, pw, od) ;
                for(uint32_t di = 0 ; di < lay_dim.in_dep ; di++)
                {
                    //filters need extra checks to make sure we are in bounds
                    for(uint16_t fh = 0 ; (fh < cw_h) ; fh++)
                    {
                        if(conv.same_pad && ((uint16_t)(fh+ph-ofs_h) >= lay_dim.hw.h)){ continue ; }  //should underflow
                        for(uint16_t fw = 0 ; (fw < cw_w) ; fw++)
                        {
                            if(conv.same_pad && ((uint16_t)(fw+pw-ofs_w) >= lay_dim.hw.w)){ continue ; }  //should underflow
                            input_i = get_input_i((fh+ph-ofs_h), (fw+pw-ofs_w), di) ;
                            filt_i = get_filter_i(fh, fw, di, od) ;
                            //multiply_f accumulate
                            if(p_tern == NULL || p_tern[filt_i] == 0)
                            {
                                if (p_filters[filt_i] == 1) {
                                    IntOps::add(&p_output->enc_segs[idx][output_i], &p_output->enc_segs[idx][output_i], &p_inputs->enc_segs[idx][input_i], st[idx][sm_ctr % sm_num]);
                                }
                                else {
                                    IntOps::subtract(&p_output->enc_segs[idx][output_i], &p_output->enc_segs[idx][output_i], &p_inputs->enc_segs[idx][input_i], st[idx][sm_ctr % sm_num]);
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "IntConv: " << time_span.count() << " seconds." << std::endl;
    //free constants input data
    fixpt_free_global((lay_dim.hw.h * lay_dim.hw.w * lay_dim.in_dep), p_inputs) ;
    return p_output ;
}

bool inline IntFunc::Convolution::retrieve_dims(uint64_t i, uint32_t ph, uint32_t pw,
                                        uint32_t* di, uint32_t* fh, uint32_t* fw)
{
    bool oob = false ;
    uint32_t cw_area = conv.window.h * conv.window.w ;
    *di = i/cw_area ;
    *fh = (i%cw_area)/conv.window.w  ;
    *fw = i%conv.window.w  ;
    //bounds check 1: make sure it is in bounds of num_ops
    if(i >= (lay_dim.in_dep*cw_area)){ oob = true ; }
    //bounds check 2: for same padding, make sure filter is in bounds
    if(conv.same_pad)
    {
        if(((uint32_t)((*fh)+ph*conv.stride.h-offset_window.h) >= lay_dim.hw.h) ||
            ((uint32_t)((*fw)+pw*conv.stride.w-offset_window.w) >= lay_dim.hw.w))
        {  oob = true ; }
    }
    return oob ;
}

uint64_t inline IntFunc::Convolution::get_input_i(uint32_t ph, uint32_t pw, uint32_t di)
{
    return (((ph)*lay_dim.hw.w + pw)*lay_dim.in_dep + di) ;
}
uint64_t inline IntFunc::Convolution::get_filter_i(uint32_t fh, uint32_t fw, uint32_t di, uint32_t od)
{
    return ((((fh)*conv.window.w + fw)*lay_dim.in_dep + di)*OutDepth + od) ;
}
uint64_t inline IntFunc::Convolution::get_output_i(uint32_t ph, uint32_t pw, uint32_t od)
{
    return (((ph)*out_hw.w + pw)*OutDepth + od) ;
}

IntFunc::SumPooling::SumPooling(tPoolParams* in_params)
{
    //error checking
    assert(in_params != NULL) ;
    assert((in_params->window.h > 0) && (in_params->window.w > 0));

    memcpy(&pool, in_params, sizeof(pool)) ;
    if(pool.stride.h == 0){ pool.stride.h = pool.window.h ; }
    if(pool.stride.w == 0){ pool.stride.w = pool.window.w ; }
}

tDimensions* IntFunc::SumPooling::prep(tDimensions* ret_dim)
{
    //input error checking
    assert(!b_prep) ;
    assert((ret_dim != NULL)) ;
    assert(((pool.window.h) != 0) && ((pool.window.w) != 0)) ;

    //copy dimension
    memcpy(&lay_dim, ret_dim, sizeof(lay_dim)) ;

    assert((pool.stride.h) != 0 && pool.stride.w != 0);

    //set output depth
    if(pool.same_pad) //round up - (partial pools used)
    {
        out_hw.h = (lay_dim.hw.h - 1)/(pool.stride.h) + 1 ;
        out_hw.w = (lay_dim.hw.w - 1)/(pool.stride.w) + 1 ;
        if(pool.stride.h == 1){ offset_window.h = (int16_t)((pool.window.h-1)/2) ; }
        else{ offset_window.h = (out_hw.h*pool.stride.h - lay_dim.hw.h)/2 ; }
        if(pool.stride.w == 1){ offset_window.w = (int16_t)((pool.window.w-1)/2) ; }
        else{ offset_window.w = (out_hw.w*pool.stride.w - lay_dim.hw.w)/2 ; }
    }
    else //valid pad - truncate (partial pools ignored)
    {
        offset_window.h = 0 ;
        offset_window.w = 0 ;
        out_hw.h = (lay_dim.hw.h-((uint16_t)(pool.window.h/2))-1)/pool.stride.h + 1 ;
        out_hw.w = (lay_dim.hw.w-((uint16_t)(pool.window.w/2))-1)/pool.stride.w + 1 ;
    }

    //update upper bound and out bits
    lay_dim.up_bound *= (pool.window.w) * (pool.window.h) ;
    for(lay_dim.out_bits = lay_dim.in_bits ;
        (lay_dim.up_bound >> lay_dim.out_bits) > 0 ;
        lay_dim.out_bits++) ;

    //update return dimensions
    ret_dim->hw.h = out_hw.h ;
    ret_dim->hw.w = out_hw.w ;
    ret_dim->in_bits = lay_dim.out_bits ;
    ret_dim->out_bits = SINGLE_BIT ;
    ret_dim->up_bound = lay_dim.up_bound  ;
    ret_dim->scale = lay_dim.scale*(pool.window.w)*(pool.window.h) ;

    b_prep = true ;

    return ret_dim ;
}

tFixedPointPacked* IntFunc::SumPooling::execute(tFixedPointPacked* p_inputs)
{
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    //input picture indexes
    tRectangle ip = {0,0} ; //input picture indexes
    uint64_t input_i, output_i ;

    //input error checking
    assert(b_prep) ;

    //allocate memory
    uint32_t len = get_size(&out_hw, SIZE_EMPTY, lay_dim.in_dep) ;
    tFixedPointPacked* p_output;
    fixpt_calloc_global(&p_output, len, 1) ;
    uint32_t opw, oph, di;
    uint32_t fh, fw;
    int idx;
    int sm_ctr = 0;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for collapse(3) shared(st) private(flag, sm_ctr, idx, output_i, input_i, opw, oph, di, fh, fw)
    for(di = 0 ; di < lay_dim.in_dep ; di++)
    {
        for(oph = 0 ; oph < out_hw.h ; oph++)
        {
            for(opw = 0 ; opw < out_hw.w; opw++)
            {
                if (flag != 1) { // should execute once per CPU thread
                  idx = omp_get_thread_num();
                  hipSetDevice(idx);
                  st[idx] = new Stream[sm_num];
                  for (int i = 0; i < sm_num; i++) {
                    st[idx][i].Create();
                  }
                  flag = 1;
                  Synchronize();
                  sm_ctr = -1;
                }
                    //get input picture indexes
                ip.h = oph * (pool.stride.h) - offset_window.h ;
                ip.w = opw * (pool.stride.w) - offset_window.w ;
                output_i = get_output_i(oph, opw, di) ;
                sm_ctr++;
                for(fh = 0 ; (fh < (pool.window.h))
                    && ((ip.h+fh) < (lay_dim.hw.h)) ; fh++)
                {
 		            if((ip.h + fh) < 0){ continue ; }
                    for(fw = 0 ; (fw < (pool.window.w))
                        && ((ip.w+fw) < lay_dim.hw.w) ; fw++)
                    {
 		                if((ip.w + fw) < 0){ continue ; }
                        input_i = get_input_i(ip.h+fh, ip.w+fw, di) ;
                        IntOps::add(&p_output->enc_segs[idx][output_i], &p_output->enc_segs[idx][output_i], &p_inputs->enc_segs[idx][input_i], st[idx][sm_ctr % sm_num]) ;
                        sm_ctr++;
                    }
                }
            }
        }
    }

    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "IntSumPool: " << time_span.count() << " seconds." << endl;
    return p_output ;
}

uint64_t inline IntFunc::SumPooling::get_input_i(uint32_t ph, uint32_t pw, uint32_t di)
{
    return (((ph)*lay_dim.hw.w + pw)*lay_dim.in_dep + di) ;
}
uint64_t inline IntFunc::SumPooling::get_output_i(uint32_t ph, uint32_t pw, uint32_t od)
{
    //indepth == outdepth
    return (((ph)*out_hw.w + pw)*lay_dim.in_dep + od) ;
}
//these functions should only be used by weight_convert program
void IntFunc::SumPooling::extract_bias(tFixedPointPacked* p_bias){ printf("Weight convert not defined\r\n") ; }

#define SLOPE_BITS 8
IntFunc::Quantize::Quantize(tQParams* qparam)
{
    dim_len = SIZE_EMPTY ;
    shift_bits = qparam->shift_bits ;
    b_prep = false ;
}


tDimensions* IntFunc::Quantize::prep(FILE* fd_bias, tDimensions* ret_dim, tMultiBitPacked* p_bias, uint16_t* p_slope)
{
    //input error checking
    assert(!b_prep) ;
    assert((ret_dim != NULL)) ;

    //get bias offset
    uint32_t bias_len = ret_dim->in_dep ;
    assert((fd_bias != NULL) && (p_bias != NULL)) ;
    BinOps::get_intfilters(fd_bias, (tMultiBitPacked**) &p_bias, bias_len) ;
    if(p_slope!=NULL && shift_bits> 1){ BinOps::get_intfilters_ptxt(fd_bias, p_slope, bias_len) ; }
    //copy dimension
    memcpy(&lay_dim, ret_dim, sizeof(lay_dim)) ;

    uint8_t sc_b = 0 ;
    for(sc_b= 0 ; (1<<sc_b) < lay_dim.scale ; sc_b++);
    slope_bits = SLOPE_BITS + sc_b - shift_bits ;

    //set output dimensions, other parameters
    if(shift_bits == 0) {
       lay_dim.out_bits = lay_dim.in_bits ;
       ret_dim->up_bound = lay_dim.up_bound ;
       ret_dim->scale = lay_dim.scale ;
    } else if(shift_bits==1) {
       lay_dim.out_bits = 1 ;
       ret_dim->up_bound = 1 ;
       ret_dim->scale = 0.5 ;
    } else {
       lay_dim.out_bits = shift_bits ;
       ret_dim->up_bound = (1<<(lay_dim.out_bits)) - 1 ;
       ret_dim->scale = ret_dim->up_bound ; 
    }
    dim_len = get_size(&lay_dim.hw, SIZE_EMPTY, lay_dim.in_dep) ;

    //update dimensions
    ret_dim->in_bits = lay_dim.out_bits ;
    ret_dim->out_bits = SINGLE_BIT ;
    ret_dim->up_bound = (1<<(lay_dim.out_bits-1)) ;
    bit_calloc_global(&p_output, dim_len);

    b_prep = true ;
    fixpt_calloc_global(&x_add, 40, 1);
    fixpt_calloc_global(&x_bn, 1, FIXEDPOINT_BITS) ;
    fixpt_calloc_global(&p_output_relu, dim_len, FIXEDPOINT_BITS);
    return ret_dim ;
}

tBitPacked* IntFunc::Quantize::execute(tFixedPointPacked* p_inputs, tFixedPointPacked* p_bias)
{
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    //input error checking
    assert(b_prep) ;

    uint32_t i;
    int idx;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    int sm_ctr = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);
    #pragma omp parallel for shared(st, x_add, p_output, p_inputs) firstprivate(sm_ctr) private(idx, flag)
    for(i = 0 ; i < dim_len ; i++)
    {
        if (flag != 1) { // should execute once per CPU thread
          idx = omp_get_thread_num();
          hipSetDevice(idx);
          st[idx] = new Stream[sm_num];
          for (int i = 0; i < sm_num; i++) {
            st[idx][i].Create();
          }
          flag = 1;
          Synchronize();
          sm_ctr = 0;
        }
        uint32_t di = i % lay_dim.in_dep ;
        IntOps::add(&x_add->enc_segs[idx][sm_ctr%sm_num], &p_inputs->enc_segs[idx][i], &p_bias->enc_segs[idx][di], st[idx][sm_ctr % sm_num]);
        IntOps::binarize_int(&x_add->enc_segs[idx][sm_ctr%sm_num].ctxt[0], st[idx][sm_ctr % sm_num]);
        Copy(p_output->enc_segs[idx][i], x_add->enc_segs[idx][sm_ctr%sm_num].ctxt[0], st[idx][sm_ctr % sm_num]);
        sm_ctr++;
    }
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "IntQuant: " << time_span.count() << " seconds.\n";
    fixpt_free_global(1, x_add) ;
    fixpt_free_global(dim_len, p_inputs) ;
    return p_output ;
}

tFixedPointPacked* IntFunc::Quantize::add_bias(tFixedPointPacked* p_inputs, tMultiBitPacked* p_bias)
{
  high_resolution_clock::time_point t1 = high_resolution_clock::now();
  tMultiBitPacked* p_output ;

  //input error checking
  assert(b_prep) ;

  //allocate memory
  mbit_calloc_global(&p_output, dim_len,1) ;
  uint32_t i;
  int idx;
  uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
  uint8_t flag = 0;
  int st_ctr = 0;
  Stream* st[NUM_GPUS];
  omp_set_num_threads(NUM_GPUS);
  #pragma omp parallel for firstprivate(st_ctr) private(idx, flag) shared(st, p_inputs, p_bias, p_output)
  for(i = 0 ; i < dim_len ; i++)
  {
      if (flag != 1) { // should execute once per CPU thread
        idx = omp_get_thread_num();
        hipSetDevice(idx);
        st[idx] = new Stream[sm_num];
        for (int i = 0; i < sm_num; i++) {
          st[idx][i].Create();
        }
        flag = 1;
      }
      uint32_t di = i % lay_dim.in_dep ;
      BinOps::int_add(p_output->enc_segs[idx][i].ctxt[0], p_inputs->enc_segs[idx][i].ctxt[0], p_bias->enc_segs[idx][di].ctxt[0], st[idx][st_ctr % sm_num]) ;
      st_ctr++;
  }
  high_resolution_clock::time_point t2 = high_resolution_clock::now();
  duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
  std::cout << "IntAddBias: " << time_span.count() << " seconds." << endl;
  mbit_free_global(dim_len, p_inputs) ;
  mbit_free_global(1, x_add) ;
  return p_output ;
}

tFixedPointPacked* IntFunc::Quantize::relu_shift(tFixedPointPacked* p_inputs, tMultiBitPacked* p_bias, uint16_t* p_slope)
{
    //input error checking
    assert(b_prep) ;
    int idx;
    uint32_t sm_num = 40; // CONFIG: set to number of streaming multiprocessors per GPU
    uint8_t flag = 0;
    int st_ctr = 0;
    Stream* st[NUM_GPUS];
    omp_set_num_threads(NUM_GPUS);

    #pragma omp parallel for firstprivate(st_ctr) private(idx, flag) shared(st, p_inputs, p_bias, p_slope, p_output_relu, slope_bits)
    for(uint64_t i = 0 ; i < dim_len ; i++)
    {
        if (flag != 1) { // should execute once per CPU thread
          idx = omp_get_thread_num();
          hipSetDevice(idx);
          st[idx] = new Stream[sm_num];
          for (int i = 0; i < sm_num; i++) {
            st[idx][i].Create();
          }
          flag = 1;
          Synchronize();
        }
        uint32_t di = i % lay_dim.in_dep ;
        IntOps::multiply_pc_ints(x_bn->enc_segs[idx][0].ctxt[0], p_inputs->enc_segs[idx][i].ctxt[0], &(p_slope[di]), (lay_dim.in_bits), SLOPE_BITS, st[idx][st_ctr % sm_num]) ;
	    IntOps::add(&(x_bn->enc_segs[idx][0]), &(x_bn->enc_segs[idx][0]), &(p_bias->enc_segs[idx][i]), st[idx][st_ctr % sm_num]) ;
        IntOps::binarize_int(&(x_bn->enc_segs[idx][0].ctxt[0]), st[idx][st_ctr % sm_num]) ;
        IntOps::shift(&(p_output_relu->enc_segs[idx][i]), &(x_bn->enc_segs[idx][0]), slope_bits, shift_bits+1, st[idx][st_ctr % sm_num]) ;
        IntOps::relu(&(p_output_relu->enc_segs[idx][i]), &(p_output_relu->enc_segs[idx][i]), MULTIBIT_BITS, st[idx][st_ctr % sm_num]) ;
	    BinOps::unbinarize_int(p_output_relu->enc_segs[idx][i].ctxt[0], st[idx][st_ctr%sm_num]);
        st_ctr++;
    }

    fixpt_free_global(1, x_bn) ;
    fixpt_free_global(dim_len, p_inputs) ;

    return p_output_relu ;
}
