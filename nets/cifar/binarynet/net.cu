#include <cstdio>
#include <cstdint>
#include <cstring>
#include <vector>
#include <thread>

#if defined(GPU_ENC)
#include "net.cuh"
#include "GPU/Layer.cuh"
#include "GPU/BinLayer.cuh"
#include "GPU/IntLayer.cuh"
#else
#include "net.h"
#include "Layer.h"
#include "BinLayer.h"
#include "IntLayer.h"
#endif
//TODO: Add in your own network file. 
//MNIST is shown here as an example
#include "../cifar.h"
//End TODO: Add in your own network file.

HeBNN::HeBNN(FILE* in_file, bool b_prep)
{
	HeBNN::init(in_file, false) ;
}

HeBNN::HeBNN()
{
	//open up input file
	FILE *fd = fopen("var_prep.dat", "r");
	if(fd == NULL)
	{
		printf("Bad Weights File. Exiting...\r\n") ;
		return ;
	}
	HeBNN::init(fd, false) ;
	fclose(fd) ;
}

void HeBNN::init(FILE* in_file, bool b_prep)
{
	tDimensions* p_dim ;
	tDimensions lay_dim ;
	uint32_t conv_dep ;
	// read eval key
	#if defined(ENCRYPTED)
	//TODO: Modify location of evaluation key if needed
	FILE* cloud_key = fopen("../../../client/eval.key","rb");
	//End TODO: Modify location of evaluation key if needed
	bk = new_tfheGateBootstrappingCloudKeySet_fromFile(cloud_key);
	fclose(cloud_key);
	#elif defined(GPU_ENC)
	//TODO: Modify location of evaluation key if needed
	ReadPubKeyFromFile(bk, "../../../client/eval.key");
	// End TODO: 
	omp_set_num_threads(NUM_GPUS);
	const auto processor_count = std::thread::hardware_concurrency();
	if (NUM_GPUS > processor_count) {
		printf("REDsec does not support configurations where the number of utilized GPUs "
					 "exceeds the number of possible CPU threads. Please reduce NUM_GPUs "
					 "in lib/GPU/Layer.cuh to reflect a value equal to or less than the "
					 "number of CPU threads in your system.");
		exit(0);
	}
	#pragma omp parallel for
	for (int i = 0; i < NUM_GPUS; i++) {
		hipSetDevice(i);
		Initialize(bk);
	}
	#else
	bk = NULL;
	#endif
	
	//model netlist
	tNetParams params ;
	//dummy params
	params.pool.window.h = 2 ;
        params.pool.window.w = 2 ;
        params.pool.stride.h = 2 ;
        params.pool.stride.w = 2 ;
        params.pool.same_pad = false ;
        params.e_bias = E_NO_BIAS ;
        conv_dep =  SIZE_EMPTY ;
        params.conv.window.h = 1 ;
        params.conv.window.w = 1 ;
        params.conv.stride.h = 1 ;
        params.conv.stride.w = 1 ;
        params.conv.tern_thresh = 0.05 ;
        params.bnorm.use_scale = false ;
        params.bnorm.eps = 0.001 ;
        params.e_bias = E_BNORM ;

	//START: Auto generated model
	  //Dimensions of input
	lay_dim.hw.h = 32 ;
	lay_dim.hw.w = 32 ;
	lay_dim.in_dep = 3 ;
	lay_dim.in_bits = 8 ;
	lay_dim.out_bits = SINGLE_BIT ;
	lay_dim.filter_bits = SINGLE_BIT ; ;
	lay_dim.bias_bits = SINGLE_BIT ;
	//Start:TODO: Update to reflect preprocessing
	  //up_bound is the range of pixel values 
	lay_dim.up_bound = 2*255 ;
	  //scale is the mulitplicitive factor to map from
	  //  floating point to integer
	lay_dim.scale = 255 ;
	//End:TODO: Update to reflect preprocessing ;


	  //Model Params and Architechture

		//Layer 0
	params.e_bias = E_NO_BIAS ;
	conv_dep =  SIZE_EMPTY ;
#if defined(GPU_ENC)
	layer0 = new IntLayer(E_NO_CONV, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params);
#else
	layer0 = new IntLayer(E_NO_CONV, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 1
	params.conv.window.h = 3 ;
	params.conv.window.w = 3 ;
	params.conv.stride.h = 1 ;
	params.conv.stride.w = 1 ;
	params.conv.same_pad = true ;
	params.conv.tern_thresh = 0.05 ;
	params.bnorm.use_scale = false ;
	params.bnorm.eps = 0.001 ;
	params.e_bias = E_BNORM ;
	conv_dep =  128 ;
#if defined(GPU_ENC)
	layer1 = new BinLayer(E_CONV, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params);
#else
	layer1 = new BinLayer(E_CONV, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 2
	params.pool.window.h = 2 ;
	params.pool.window.w = 2 ;
	params.pool.stride.h = 2 ;
	params.pool.stride.w = 2 ;
	params.pool.same_pad = false ;
	conv_dep =  128 ;
#if defined(GPU_ENC)
	layer2 = new BinLayer(E_CONV, conv_dep, E_MAXPOOL, E_ACTIVATION_SIGN, &params);
#else
	layer2 = new BinLayer(E_CONV, conv_dep, E_MAXPOOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 3
	conv_dep =  256 ;
#if defined(GPU_ENC)
	layer3 = new BinLayer(E_CONV, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params);
#else
	layer3 = new BinLayer(E_CONV, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 4
	conv_dep =  256 ;
#if defined(GPU_ENC)
	layer4 = new BinLayer(E_CONV, conv_dep, E_MAXPOOL, E_ACTIVATION_SIGN, &params);
#else
	layer4 = new BinLayer(E_CONV, conv_dep, E_MAXPOOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 5
	conv_dep =  512 ;
#if defined(GPU_ENC)
	layer5 = new BinLayer(E_CONV, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params);
#else
	layer5 = new BinLayer(E_CONV, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 6
	conv_dep =  512 ;
#if defined(GPU_ENC)
	layer6 = new BinLayer(E_CONV, conv_dep, E_MAXPOOL, E_ACTIVATION_SIGN, &params);
#else
	layer6 = new BinLayer(E_CONV, conv_dep, E_MAXPOOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 7
	params.conv.window.h = 1 ;
	params.conv.window.w = 1 ;
	conv_dep =  1024 ;
#if defined(GPU_ENC)
	layer7 = new BinLayer(E_FC, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params);
#else
	layer7 = new BinLayer(E_FC, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 8
	conv_dep =  1024 ;
#if defined(GPU_ENC)
	layer8 = new BinLayer(E_FC, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params);
#else
	layer8 = new BinLayer(E_FC, conv_dep, E_NO_POOL, E_ACTIVATION_SIGN, &params, bk) ;
#endif

		//Layer 9
	params.e_bias = E_NO_BIAS ;
	conv_dep =  10 ;
#if defined(GPU_ENC)
	layer9 = new BinLayer(E_FC, conv_dep, E_NO_POOL, E_ACTIVATION_NONE, &params);
#else
	layer9 = new BinLayer(E_FC, conv_dep, E_NO_POOL, E_ACTIVATION_NONE, &params, bk) ;
#endif
	//END: Auto generated model
	
	//extract weights and initialize the dimensions of the model
	//START: Model prep
	p_dim = &lay_dim ;
	p_dim = (tDimensions*) layer0->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer1->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer2->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer3->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer4->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer5->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer6->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer7->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer8->prep(in_file, p_dim) ;
	p_dim = (tDimensions*) layer9->prep(in_file, p_dim) ;
	//END: Model prep

}


#if defined(ENCRYPTED)
tMultiBit* HeBNN::run(tMultiBit* in_data)
#elif defined(GPU_ENC)
tMultiBitPacked* HeBNN::run(tMultiBitPacked* in_data)
#else
tMultiBit* HeBNN::run(tFixedPoint* in_data)
#endif
{
	#if defined(GPU_ENC)
	tMultiBitPacked* mbdata ;
	tBitPacked* bdata ;
	#else
	tBit* bdata ;
	tMultiBit* mbdata ;
	#endif

	//START: Execute inference model
#if !defined(GPU_ENC)
	bdata = (tBit*)layer0->execute(in_data) ;
#else
	bdata = (tBitPacked*)layer0->execute(in_data) ;
#endif
#if !defined(GPU_ENC)
	bdata = (tBit*)layer1->execute((tBit*) bdata) ;
#else
	bdata = (tBitPacked*)layer1->execute((tBitPacked*) bdata) ;
#endif
#if !defined(GPU_ENC)
	bdata = (tBit*)layer2->execute((tBit*) bdata) ;
#else
	bdata = (tBitPacked*)layer2->execute((tBitPacked*) bdata) ;
#endif
#if !defined(GPU_ENC)
	bdata = (tBit*)layer3->execute((tBit*) bdata) ;
#else
	bdata = (tBitPacked*)layer3->execute((tBitPacked*) bdata) ;
#endif
#if !defined(GPU_ENC)
	bdata = (tBit*)layer4->execute((tBit*) bdata) ;
#else
	bdata = (tBitPacked*)layer4->execute((tBitPacked*) bdata) ;
#endif
#if !defined(GPU_ENC)
	bdata = (tBit*)layer5->execute((tBit*) bdata) ;
#else
	bdata = (tBitPacked*)layer5->execute((tBitPacked*) bdata) ;
#endif
#if !defined(GPU_ENC)
	bdata = (tBit*)layer6->execute((tBit*) bdata) ;
#else
	bdata = (tBitPacked*)layer6->execute((tBitPacked*) bdata) ;
#endif
#if !defined(GPU_ENC)
	bdata = (tBit*)layer7->execute((tBit*) bdata) ;
#else
	bdata = (tBitPacked*)layer7->execute((tBitPacked*) bdata) ;
#endif
#if !defined(GPU_ENC)
	bdata = (tBit*)layer8->execute((tBit*) bdata) ;
#else
	bdata = (tBitPacked*)layer8->execute((tBitPacked*) bdata) ;
#endif
#if !defined(GPU_ENC)
	mbdata = (tMultiBit*)layer9->execute((tBit*) bdata) ;
#else
	mbdata = (tMultiBitPacked*)layer9->execute((tBitPacked*) bdata) ;
#endif
	//END: Execute inference model
	
	#if defined(GPU_ENC)
	return (tMultiBitPacked*)mbdata ;
	#else
	return (tMultiBit*)mbdata ;
	#endif
}

#ifdef _WEIGHT_CONVERT_
void HeBNN::export_weights(FILE* out_file)
{
	//START: Weight convert
layer0->export_weights(out_file) ;
layer1->export_weights(out_file) ;
layer2->export_weights(out_file) ;
layer3->export_weights(out_file) ;
layer4->export_weights(out_file) ;
layer5->export_weights(out_file) ;
layer6->export_weights(out_file) ;
layer7->export_weights(out_file) ;
layer8->export_weights(out_file) ;
layer9->export_weights(out_file) ;
	//END: Weight convert
	
	print_status("Exported Success\r\n") ;

	return ;
}
#else
void export_weights(FILE* out_file){ printf("Weight Convert not defined\r\n") ; }
#endif


void HeBNN::get_in_dims(tDimensions* in)
{
	if(in != NULL)
	{
		memcpy(in, &(layer0->in_dim), sizeof(*in)) ;
	}
}

void HeBNN::get_out_dims(tDimensions* out)
{
	if(out != NULL)
	{
		//START: Get out dimensions
		memcpy(out, &(layer9->out_dim), sizeof(*out)) ;
		//END: Get out dimensions
	}
}

