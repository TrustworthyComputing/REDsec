#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include "../mnist.h"

#if defined(GPU_ENC)
#include "../../../lib/GPU/Layer.cuh"
#include "net.cuh"
#include "REDcuFHE/redcufhe_gpu.cuh"
#include <ctime>
#include <ratio>
#include <chrono>
using namespace redcufhe;
using namespace std::chrono;
#else
#include "../../../lib/Layer.h"
#include "net.h"
#endif


/*********************************** Defines ************************************/
#define CSV_FIELD_LEN 4 //3digit number with a comma seperator
#define NUM_SAMPLES    ((uint32_t)1)
#define UPDATES     ((uint32_t)(NUM_SAMPLES/10.0+.999))
/****************************** Structs And Enums *******************************/
typedef eMnist eResult ;
/**************************** Function Declarations *****************************/
#ifndef ENCRYPTED
eResult findResult(tFixedPoint* data, tDimensions* dim) ;
static tFixedPoint* convert_line(char* line, tDimensions* dim, eResult* p_label) ;
#endif
void reset_dim(tDimensions* dim) ;

int main(void)
{
    //dimensions
    tDimensions indim ;
    tDimensions outdim ;

    //create network
    #if defined(GPU_ENC)
    hipSetDevice(0);
    #endif 
    print_status("Instantiating network architecture...\n");
    HeBNN* network = new HeBNN() ;
    network->get_in_dims(&indim) ;
    network->get_out_dims(&outdim) ;

    //csv input
    FILE *fd_image = NULL ;

    //Results
    #if defined(GPU_ENC)
    size_t in_size = indim.hw.h * indim.hw.w * indim.in_dep;
    tMultiBitPacked* nn_data;
    mbit_calloc_global(&nn_data, in_size, 8);
    tMultiBitPacked* nn_result ;
    mbit_calloc_global(&nn_result, outdim.in_dep, 1);

    // Read encrypted image
    for (int k = 0; k < NUM_GPUS; k++) {
      hipSetDevice(k);
      std::ifstream input_file("../../../client/image.ctxt");
      for(uint32_t i = 0; i < in_size; i++) {
        for (uint8_t j = 0; j < 8; j++) {
          ReadCtxtFromFileRed(nn_data->enc_segs[k][i].ctxt[j], input_file);
        }
      }
      input_file.close();
    }
    printf("Running network...\n");
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    nn_result = network->run(nn_data);
    Synchronize();
    CuCheckError();
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Inference Time: " << time_span.count() << " seconds\n";
    CleanUp();
    exit(0);
    #elif defined(ENCRYPTED)
    size_t in_size = indim.hw.h * indim.hw.w * indim.in_dep ;
    tMultiBit* nn_data = new tMultiBit[in_size];
    for (uint32_t i = 0; i < in_size; i++) {
      nn_data[i].size = 1;
      nn_data[i].ctxt = new_gate_bootstrapping_ciphertext_array(nn_data[i].size, network->bk->params);
    }
    tFixedPoint* nn_result ;
    #else
    size_t line_size = (CSV_FIELD_LEN * (indim.hw.h) * (indim.hw.w) * (indim.in_dep)) ;
    char* file_line = (char*) malloc(line_size) ;
    tFixedPoint* nn_data ;
    uint32_t image_i = 0 ;
    uint32_t correct = 0 ;
    eResult label = RESULT_ERROR ;
    eResult result = RESULT_ERROR ;
    tFixedPoint* nn_result ;
    #endif

    //Open File
    #if defined(ENCRYPTED)
    fd_image = fopen("../../../client/image.ctxt", "rb");
    #elif !defined(GPU_ENC)
    fd_image = fopen("../mnist_test.csv", "r");
    #endif

    #if !defined(GPU_ENC)
    if(fd_image == NULL)
    {
        printf("Bad Sample File. Exiting...\r\n") ;
        return -1 ;
    }
    #endif

    #ifdef ENCRYPTED //only one image
    for(uint32_t i = 0; i < in_size; i++) {
      import_gate_bootstrapping_ciphertext_fromFile(fd_image, &nn_data[i].ctxt[0], network->bk->params);
    }
    printf("Running network...\n");
    nn_result = network->run(nn_data);
    FILE* output_ctxts = fopen("../../../client/network_output.ctxt", "wb");

    // output result to file
    for (uint8_t i = 0; i < outdim.in_dep; i++) {
      export_gate_bootstrapping_ciphertext_toFile(output_ctxts, &nn_result[i].ctxt[0], network->bk->params);
    }

    printf("Result ctxts loaded into network_output.ctxt.\n");
    fclose(output_ctxts);

    // clean up ctxt
    for (uint32_t i = 0; i < outdim.in_dep; i++) {
      delete_gate_bootstrapping_ciphertext_array(nn_result[i].size, nn_result[i].ctxt);
    }
    free(nn_result);
    #else
    //Run network on images
    for(image_i = 0 ; image_i < NUM_SAMPLES ; image_i++)
    {
      label = RESULT_ERROR ;
      result = RESULT_ERROR ;
      //get line
      file_line = fgets(file_line, line_size, fd_image) ;
      if(*file_line < '0' || *file_line>'9'){ continue ; }
      //convert data to appropriate struct
      nn_data = convert_line(file_line, &indim, &label) ;
      //Run NN
      nn_result = (tFixedPoint*) network->run(nn_data) ;
      //Get result
      result = findResult(nn_result, &outdim) ;
      if(result == label){ correct++ ; }
      if((image_i % UPDATES)==0)
      {
        printf("correct: %4d\timage_i: %4d\tLabel: %d\tPrediction: %d\r\n", correct, image_i, label, result) ;
      }
      free(nn_result) ;
    }
    printf("Correct: %2f%%\r\n", ((float)correct*100.0)/image_i) ;
    #endif
    fclose(fd_image) ;

    #ifdef ENCRYPTED
    delete_gate_bootstrapping_cloud_keyset(network->bk);
    #endif
}

#if !defined(ENCRYPTED)
eResult findResult(tFixedPoint* data, tDimensions* dim)
{
  tFixedPoint val = 0 ;
  eResult res = RESULT_ERROR ;

  if(data != NULL)
  {
    val = data[0];
    res = CAT_0 ;
    for(uint16_t image_i = (int) CAT_0 ; image_i < (dim->in_dep) ; image_i++)
    {
      if(data[image_i] > val)
      {
        res = (eResult) image_i ;
        val = data[image_i] ;
      }
    }
  }
  return res ;
}

//TODO: THis is the function preprocessing
static tFixedPoint* convert_line(char* line, tDimensions* dim, eResult* p_label)
{
  uint32_t len = get_size(&(dim->hw), dim->in_dep, SIZE_EMPTY) ;
  tFixedPoint* output = (tFixedPoint*) calloc(len, sizeof(tFixedPoint)) ;
  char* tok = strtok(line, ",") ; //first number is the label
  *p_label = (eResult) atoi(tok) ;
  for (uint32_t i = 0 ; i < len ; i++)
  {
    tok = strtok(NULL, ",\n") ;
    if((tok != NULL) && (*tok != '\0'))
    {
      output[i] = (tFixedPoint) (2*atoi(tok)-255) ;
    }
  }
  return output ;
}
#endif
