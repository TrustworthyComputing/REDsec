#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <fstream>
#include <iostream>
#include "../../../lib/GPU/Layer.cuh"
#include "net.cuh"
#include "../mnist.h"
#include <ctime>
#include <ratio>
#include <chrono>

#include "REDcuFHE/redcufhe_gpu.cuh"

/*********************************** Defines ************************************/
#define CSV_FIELD_LEN 4 //3digit number with a comma seperator
#define NUM_SAMPLES    ((uint32_t)100)
#define UPDATES     ((uint32_t)(NUM_SAMPLES/10.0+.999))
/****************************** Structs And Enums *******************************/
typedef eMnist eResult ;
/**************************** Function Declarations *****************************/
void reset_dim(tDimensions* dim) ;

using namespace redcufhe;
using namespace std::chrono;

int main(void)
{
    //dimensions
    tDimensions indim ;
    tDimensions outdim ;
    //create network
    hipSetDevice(0);
    print_status("Instantiating network architecture...\n");
    HeBNN* network = new HeBNN() ;
    network->get_in_dims(&indim) ;
    network->get_out_dims(&outdim) ;

    //Results
    size_t in_size = indim.hw.h * indim.hw.w * indim.in_dep;
    tMultiBitPacked* nn_data;
    mbit_calloc_global(&nn_data, in_size, 8);
    tMultiBitPacked* nn_result ;
    mbit_calloc_global(&nn_result, 10, 1);

    // Read encrypted image
    for (int k = 0; k < NUM_GPUS; k++) {
      hipSetDevice(k);
      std::ifstream input_file("../../../client/image.ctxt");
      for(uint32_t i = 0; i < in_size; i++) {
        for (uint8_t j = 0; j < 8; j++) {
          ReadCtxtFromFileRed(nn_data->enc_segs[k][i].ctxt[j], input_file);
        }
      }
      input_file.close();
    }

    printf("Running network...\n");
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    nn_result = network->run(nn_data);
    Synchronize();
    CuCheckError();
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Inference Time: " << time_span.count() << " seconds\n";
    CleanUp();
}
